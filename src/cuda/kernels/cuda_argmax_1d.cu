#include <hipcub/hipcub.hpp>
#include <cub/device/device_reduce.cuh>

extern "C"
void cuda_argmax_1d( float* input, int n_input, float* cache, int n_cache, int* result, hipStream_t sm )
{
    std::size_t cache_size = static_cast<std::size_t>(n_cache * sizeof(float));
    hipcub::KeyValuePair<int, float> *_result = reinterpret_cast<hipcub::KeyValuePair<int, float>*>(result);
    //hipcub::DeviceReduce::ArgMax( cache, cache_size, input,  _result, n_input, sm, false );
    hipcub::DeviceReduce::ArgMax( cache, cache_size, input,  _result, n_input, sm );
}

